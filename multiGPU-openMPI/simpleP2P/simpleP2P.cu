/*
 * Copyright 1993-2014 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
 * This sample demonstrates a combination of Peer-to-Peer (P2P) and
 * Unified Virtual Address Space (UVA) features new to SDK 4.0
 */

// includes, system

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

__global__ void SimpleKernel(float *src, float *dst)
{
    // Just a dummy kernel, doing enough for us to verify that everything
    // worked
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    dst[idx] = src[idx] * 2.0f;
}

inline bool IsGPUCapableP2P(hipDeviceProp_t *pProp)
{
#ifdef _WIN32
    return (bool)(pProp->tccDriver ? true : false);
#else
    return (bool)(pProp->major >= 2);
#endif
}

inline bool IsAppBuiltAs64()
{
#if defined(__x86_64) || defined(AMD64) || defined(_M_AMD64)
    return 1;
#else
    return 0;
#endif
}

int main(int argc, char **argv)
{
    printf("[%s] - Starting...\n", argv[0]);

    if (!IsAppBuiltAs64())
    {
        printf("%s is only supported with on 64-bit OSs and the application must be built as a 64-bit target.  Test is being waived.\n", argv[0]);
    }

    // Number of GPUs
    printf("Checking for multiple GPUs...\n");
    int gpu_n;
    hipGetDeviceCount(&gpu_n);
    printf("CUDA-capable device count: %i\n", gpu_n);

    if (gpu_n < 2)
    {
        printf("Two or more GPUs with SM 2.0 or higher capability are required for %s.\n", argv[0]);
        printf("Waiving test.\n");
    }

    // Query device properties
    hipDeviceProp_t prop[64];
    int gpuid[64]; // we want to find the first two GPU's that can support P2P
    int gpu_count = 0;   // GPUs that meet the criteria

    for (int i=0; i < gpu_n; i++)
    {
        hipGetDeviceProperties(&prop[i], i);

        // Only boards based on Fermi can support P2P
        if ((prop[i].major >= 2)
#ifdef _WIN32
            // on Windows (64-bit), the Tesla Compute Cluster driver for windows must be enabled
            && prop[i].tccDriver
#endif
           )
        {
            // This is an array of P2P capable GPUs
            gpuid[gpu_count++] = i;
        }

        printf("> GPU%d = \"%15s\" %s capable of Peer-to-Peer (P2P)\n", i, prop[i].name, (IsGPUCapableP2P(&prop[i]) ? "IS " : "NOT"));
    }

    // Check for TCC for Windows
    if (gpu_count < 2)
    {
        printf("\nThis sample requires two SM 2.0 GPUs to use P2P/UVA functionality.\n");
#ifdef _WIN32
        printf("\nFor Windows Vista/Win7, a TCC driver must be installed and enabled to use P2P/UVA functionality.\n");
#endif
        hipSetDevice(0);

        // cudaDeviceReset causes the driver to clean up all state. While
        // not mandatory in normal operation, it is good practice.  It is also
        // needed to ensure correct operation when the application is being
        // profiled. Calling cudaDeviceReset causes all profile data to be
        // flushed before the application exits
        hipDeviceReset();
        exit(EXIT_SUCCESS);
    }

#if CUDART_VERSION >= 4000
    // Check possibility for peer access
    printf("\nChecking GPU(s) for support of peer to peer memory access...\n");
    int can_access_peer_0_1, can_access_peer_1_0;
    // In this case we just pick the first two that we can support
    hipDeviceCanAccessPeer(&can_access_peer_0_1, gpuid[0], gpuid[1]);
    hipDeviceCanAccessPeer(&can_access_peer_1_0, gpuid[1], gpuid[0]);

    // Output results from P2P capabilities
    printf("> Peer-to-Peer (P2P) access from %s (GPU%d) -> %s (GPU%d) : %s\n", prop[gpuid[0]].name, gpuid[0],
           prop[gpuid[1]].name, gpuid[1] ,
           can_access_peer_0_1 ? "Yes" : "No");
    printf("> Peer-to-Peer (P2P) access from %s (GPU%d) -> %s (GPU%d) : %s\n", prop[gpuid[1]].name, gpuid[1],
           prop[gpuid[0]].name, gpuid[0],
           can_access_peer_1_0 ? "Yes" : "No");

    if (can_access_peer_0_1 == 0 || can_access_peer_1_0 == 0)
    {
        printf("Two or more SM 2.0 class GPUs are required for %s to run.\n", argv[0]);
        printf("Support for UVA requires a GPU with SM 2.0 capabilities.\n");
        printf("Peer to Peer access is not available between GPU%d <-> GPU%d, waiving test.\n", gpuid[0], gpuid[1]);
        hipSetDevice(gpuid[0]);

        // cudaDeviceReset causes the driver to clean up all state. While
        // not mandatory in normal operation, it is good practice.  It is also
        // needed to ensure correct operation when the application is being
        // profiled. Calling cudaDeviceReset causes all profile data to be
        // flushed before the application exits
        hipDeviceReset();
        hipSetDevice(gpuid[1]);
        hipDeviceReset();
        exit(EXIT_SUCCESS);
    }

    // Enable peer access
    printf("Enabling peer access between GPU%d and GPU%d...\n", gpuid[0], gpuid[1]);
    hipSetDevice(gpuid[0]);
    hipDeviceEnablePeerAccess(gpuid[1], 0);
    hipSetDevice(gpuid[1]);
    hipDeviceEnablePeerAccess(gpuid[0], 0);

    // Check that we got UVA on both devices
    printf("Checking GPU%d and GPU%d for UVA capabilities...\n", gpuid[0], gpuid[1]);
    const bool has_uva = (prop[gpuid[0]].unifiedAddressing && prop[gpuid[1]].unifiedAddressing);

    printf("> %s (GPU%d) supports UVA: %s\n", prop[gpuid[0]].name, gpuid[0], (prop[gpuid[0]].unifiedAddressing ? "Yes" : "No"));
    printf("> %s (GPU%d) supports UVA: %s\n", prop[gpuid[1]].name, gpuid[1], (prop[gpuid[1]].unifiedAddressing ? "Yes" : "No"));

    if (has_uva)
    {
        printf("Both GPUs can support UVA, enabling...\n");
    }
    else
    {
        printf("At least one of the two GPUs does NOT support UVA, waiving test.\n");
        exit(EXIT_SUCCESS);
    }

    // Allocate buffers
    const size_t buf_size = 1024 * 1024 * 16 * sizeof(float);
    printf("Allocating buffers (%iMB on GPU%d, GPU%d and CPU Host)...\n", int(buf_size / 1024 / 1024), gpuid[0], gpuid[1]);
    hipSetDevice(gpuid[0]);
    float *g0;
    hipMalloc(&g0, buf_size);
    hipSetDevice(gpuid[1]);
    float *g1;
    hipMalloc(&g1, buf_size);
    float *h0;
    hipHostMalloc(&h0, buf_size, hipHostMallocDefault); // Automatically portable with UVA

    // Create CUDA event handles
    printf("Creating event handles...\n");
    hipEvent_t start_event, stop_event;
    float time_memcpy;
    int eventflags = hipEventBlockingSync;
    hipEventCreateWithFlags(&start_event, eventflags);
    hipEventCreateWithFlags(&stop_event, eventflags);

    // P2P memcopy() benchmark
    hipEventRecord(start_event, 0);

    for (int i=0; i<100; i++)
    {
        // With UVA we don't need to specify source and target devices, the
        // runtime figures this out by itself from the pointers

        // Ping-pong copy between GPUs
        if (i % 2 == 0)
        {
            hipMemcpy(g1, g0, buf_size, hipMemcpyDefault);
        }
        else
        {
            hipMemcpy(g0, g1, buf_size, hipMemcpyDefault);
        }
    }

    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&time_memcpy, start_event, stop_event);
    printf("hipMemcpyPeer / hipMemcpy between GPU%d and GPU%d: %.2fGB/s\n", gpuid[0], gpuid[1],
           (1.0f / (time_memcpy / 1000.0f)) * ((100.0f * buf_size)) / 1024.0f / 1024.0f / 1024.0f);

    // Prepare host buffer and copy to GPU 0
    printf("Preparing host buffer and memcpy to GPU%d...\n", gpuid[0]);

    for (int i=0; i<buf_size / sizeof(float); i++)
    {
        h0[i] = float(i % 4096);
    }

    hipSetDevice(gpuid[0]);
    hipMemcpy(g0, h0, buf_size, hipMemcpyDefault);

    // Kernel launch configuration
    const dim3 threads(512, 1);
    const dim3 blocks((buf_size / sizeof(float)) / threads.x, 1);

    // Run kernel on GPU 1, reading input from the GPU 0 buffer, writing
    // output to the GPU 1 buffer
    printf("Run kernel on GPU%d, taking source data from GPU%d and writing to GPU%d...\n",
           gpuid[1], gpuid[0], gpuid[1]);
    hipSetDevice(gpuid[1]);
    SimpleKernel<<<blocks, threads>>>(g0, g1);

    hipDeviceSynchronize();

    // Run kernel on GPU 0, reading input from the GPU 1 buffer, writing
    // output to the GPU 0 buffer
    printf("Run kernel on GPU%d, taking source data from GPU%d and writing to GPU%d...\n",
           gpuid[0], gpuid[1], gpuid[0]);
    hipSetDevice(gpuid[0]);
    SimpleKernel<<<blocks, threads>>>(g1, g0);

    hipDeviceSynchronize();

    // Copy data back to host and verify
    printf("Copy data back to host from GPU%d and verify results...\n", gpuid[0]);
    hipMemcpy(h0, g0, buf_size, hipMemcpyDefault);

    int error_count = 0;

    for (int i=0; i<buf_size / sizeof(float); i++)
    {
        // Re-generate input data and apply 2x '* 2.0f' computation of both
        // kernel runs
        if (h0[i] != float(i % 4096) * 2.0f * 2.0f)
        {
            printf("Verification error @ element %i: val = %f, ref = %f\n", i, h0[i], (float(i%4096)*2.0f*2.0f));

            if (error_count++ > 10)
            {
                break;
            }
        }
    }

    // Disable peer access (also unregisters memory for non-UVA cases)
    printf("Enabling peer access...\n");
    hipSetDevice(gpuid[0]);
    hipDeviceDisablePeerAccess(gpuid[1]);
    hipSetDevice(gpuid[1]);
    hipDeviceDisablePeerAccess(gpuid[0]);

    // Cleanup and shutdown
    printf("Shutting down...\n");
    hipEventDestroy(start_event);
    hipEventDestroy(stop_event);
    hipSetDevice(gpuid[0]);
    hipFree(g0);
    hipSetDevice(gpuid[1]);
    hipFree(g1);
    hipHostFree(h0);

    for (int i=0; i<gpu_n; i++)
    {
        hipSetDevice(i);

        // cudaDeviceReset causes the driver to clean up all state. While
        // not mandatory in normal operation, it is good practice.  It is also
        // needed to ensure correct operation when the application is being
        // profiled. Calling cudaDeviceReset causes all profile data to be
        // flushed before the application exits
        hipDeviceReset();
    }

    if (error_count != 0)
    {
        printf("Test failed!\n");
        exit(EXIT_FAILURE);
    }
    else
    {
        printf("Test passed\n");
        exit(EXIT_SUCCESS);
    }

#else // Using CUDA 3.2 or older
    printf("simpleP2P requires CUDA 4.0 to build and run, waiving testing\n");
    exit(EXIT_SUCCESS);
#endif

}